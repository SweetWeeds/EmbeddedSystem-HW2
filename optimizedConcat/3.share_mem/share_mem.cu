#include "user.h"

using namespace std;

int main(int argc, char *argv[]) {
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<int> dis(1, 10);
    hipEvent_t cuda_start, cuda_end;
    float host_exec_time, device_exec_time;

    printf("mat1[%d][%d]\n", MAT1_ROW, MAT1_COL);
    printf("mat2[%d][%d]\n", MAT2_ROW, MAT2_COL);
    printf("matr[%d][%d]\n", MATR_ROW, MATR_COL);

    int mat1_size = MAT1_ROW * MAT1_COL * sizeof(int);
    int mat2_size = MAT2_ROW * MAT2_COL * sizeof(int);
    int matr_size = MATR_ROW * MATR_COL * sizeof(int);
    
    int *host_mat1 = NULL, *host_mat2 = NULL, *host_matr = NULL;
    int *device_mat1 = NULL, *device_mat2 = NULL, *device_matr = NULL;
    int *host_device_matr = NULL;
    
    // Start of Memory Allocation //
    host_mat1 = (int *)malloc(mat1_size);
    host_mat2 = (int *)malloc(mat2_size);
    host_matr = (int *)malloc(matr_size);
    host_device_matr = (int *)malloc(matr_size);
    hipMalloc((void **)&device_mat1, mat1_size);
    hipMalloc((void **)&device_mat2, mat2_size);
    hipMalloc((void **)&device_matr, matr_size);
    // End of Memory Allocation //

    // Start of Array Initalization //
    // Matrix 1
    for (int i = 0; i < MAT1_ROW; i++) {
        for (int j = 0; j < MAT1_COL; j++) {
            host_mat1[i * MAT1_COL + j] = dis(gen);
        }
    }

    // Matrix 2
    for (int i = 0; i < MAT2_ROW; i++) {
        for (int j = 0; j < MAT2_COL; j++) {
            host_mat2[i * MAT2_COL + j] = dis(gen);
        }
    }
    // End of Array Initalization //

    // Memory Copy (Matrix 1, 2)
    hipMemcpy(device_mat1, host_mat1, mat1_size, hipMemcpyHostToDevice);
    hipMemcpy(device_mat2, host_mat2, mat2_size, hipMemcpyHostToDevice);

    // Prepare File Stream
    ofstream resultFile;
    resultFile.open("./result.csv", ios::out);
    resultFile << "Target,#Threads,#ThreadBlks,ExecTime\n";

    // Start of Concatenation (host) //
    struct timeval startTime, endTime;
    gettimeofday(&startTime, NULL);
    host_Concatenate(host_mat1, host_mat2, host_matr, &host_exec_time);
    gettimeofday(&endTime, NULL);
    host_exec_time = (endTime.tv_sec - startTime.tv_sec) * 1000. + (endTime.tv_usec - startTime.tv_usec) / 1000.;
    resultFile << "host,1,1," << host_exec_time << "\n";    // Write information of execution on host.
    printf("[INFO] Host Execution time:%lf\n", host_exec_time);
    // End of Concatenation (host) //

    // Start of Concatenation (device) //
    int numElements = MATR_COL * MATR_ROW;
    int numBlocks, numThreadsperBlock;
    for (numThreadsperBlock = NUM_THREADS_BASE; numThreadsperBlock <= NUM_THREADSA_MAX; numThreadsperBlock *= 2) {
        for (numBlocks = NUM_THREAD_BLKS_FROM; numBlocks <= NUM_THREAD_BLKS_TO; numBlocks *= 2) {
            int numOps = numElements > (numBlocks * numThreadsperBlock) ?
                        numElements / (numBlocks * numThreadsperBlock) + (numElements % (numBlocks * numThreadsperBlock) ? 1 : 0) : 1;
            dim3 gridSize(numBlocks);
            dim3 blockSize(numThreadsperBlock);
            float tmp_exec_time;
            device_exec_time = 0.0;

            for (int i = 0; i < ITERATION; i++) {
                hipMemset(device_matr, 0, numElements);    // Initialize values of 'device_matr'
                hipEventCreate(&cuda_start);
                hipEventCreate(&cuda_end);
                hipEventRecord(cuda_start, 0);
                device_Concatenate<<<gridSize, blockSize, numThreadsperBlock>>>(device_mat1, device_mat2, device_matr,
                                                            numOps, numElements, MAT1_COL, MAT2_COL, MATR_COL);
                hipEventRecord(cuda_end, 0);
                hipEventSynchronize(cuda_end);
                hipEventElapsedTime(&tmp_exec_time, cuda_start, cuda_end);
                hipEventDestroy(cuda_start);
                hipEventDestroy(cuda_end);
                device_exec_time += tmp_exec_time;
            }
            device_exec_time /= ITERATION;

            // Write information of execution on device.
            resultFile << "device," << numThreadsperBlock << "," << numBlocks << "," << device_exec_time << "\n";
            hipMemcpy(host_device_matr, device_matr, matr_size, hipMemcpyDeviceToHost);
            // Compare concatenation results of host and device
            int diff = compareArray(host_matr, host_device_matr, numElements);
            if (diff) {
                printf("[WARNING] ");   // Different points are exist.
            } else {
                printf("[INFO] ");      //
            }
            printf("numOps: %d, numBlocks: %d, numThreadsperBlock: %d, diff: %d, exec_time: %.4lf\n", numOps, numBlocks, numThreadsperBlock, diff, device_exec_time);
        }
    }
    // End of Concatenation (device) //

    free(host_mat1);
    free(host_mat2);
    free(host_matr);
    free(host_device_matr);
    hipFree(device_mat1);
    hipFree(device_mat2);
    hipFree(device_matr);

    resultFile.close();     // Close File Stream

    return 0;
}
