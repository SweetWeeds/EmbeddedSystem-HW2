#include "hip/hip_runtime.h"
#include "user_device.h"

__global__
void device_Concatenate(int *device_mat1, int *device_mat2, int *device_matr,
                        int numOps, int numElements, int mat1_col, int mat2_col, int matr_col) {
    __shared__ int subMatr[1024];
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int row_index, col_index;
    int offset = gridDim.x * blockDim.x;

    for (int i = 0; i < numOps; i++) {
        row_index = index / matr_col;
        col_index = index % matr_col;
        if (index < numElements) {
            subMatr[threadIdx.x] = (col_index < mat1_col) ?
                device_mat1[row_index * mat1_col + col_index] :
                device_mat2[row_index * mat2_col + (col_index - mat1_col)];
            //__syncthreads();
            device_matr[index] = subMatr[threadIdx.x];
            index += offset;
        }
    }
}
