#include "hip/hip_runtime.h"
#include "user_device.h"

__global__
void device_Concatenate(int *device_mat1, int *device_mat2, int* device_matr,
                        int numOps, int numElements, int mat1_col, int mat2_col, int matr_col) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int row_index, col_index;
    int offset = gridDim.x * blockDim.x;

    for (int i = 0; i < numOps; i++) {
        row_index = index / matr_col;
        col_index = index % matr_col;
        if (index < numElements) {
            device_matr[index] = (col_index < mat1_col) ?
                device_mat1[row_index * mat1_col + col_index] :
                device_mat2[row_index * mat2_col + (col_index - mat1_col)];
            index += offset;
        }
    }
}
